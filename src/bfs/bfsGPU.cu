#include "hip/hip_runtime.h"
#include "bfsGPU.hpp"
#include <chrono>
#include <cstdio>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
namespace bfsGPU {

	const int BLOCK_SIZE = 1024;
	const int BLOCK_QUEUE_SIZE = 32;
	const int SUB_QUEUE_SIZE = 4;
	const int NUM_SUB_QUEUES = 32;

	//device pointers
	int *d_adjList;
	int *d_edgeOffsets;
	int *d_vertexDegree;
	int *d_distance;
	int *d_parent;
	int *d_currQ;
	int *d_nextQ;
	int *d_nextQSize;
	texture<int, hipTextureType1D, hipReadModeElementType> tex_edgeOffsets;


	void initMemory(Graph &G, int source, std::vector<int> &distanceCheck) {

		//Allocation
		hipMalloc(&d_adjList, G.numEdges_m * sizeof(int));
		hipMalloc(&d_edgeOffsets, G.numVertices_m * sizeof(int));
		hipMalloc(&d_vertexDegree, G.numVertices_m * sizeof(int));
		hipMalloc(&d_distance, G.numVertices_m * sizeof(int));
		hipMalloc(&d_currQ, G.numVertices_m * sizeof(int));
		hipMalloc(&d_nextQ, G.numVertices_m * sizeof(int));
		hipMalloc(&d_nextQSize, sizeof(int));
		//Data transfer
		hipMemcpy(d_adjList, G.adjacencyList_m.data(), G.numEdges_m * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_edgeOffsets, G.edgeOffsets_m.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_vertexDegree, G.vertexDegree_m.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);
		//Init kernel parameters
		hipMemcpy(d_currQ, &source, sizeof(int), hipMemcpyHostToDevice);
		hipMemset(d_nextQSize, 0, sizeof(int));
		//Init distance
		distanceCheck.resize(G.numVertices_m);
		std::fill(distanceCheck.begin(), distanceCheck.end(), -1);
		distanceCheck[source] = 0;
		hipMemcpy(d_distance, distanceCheck.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);
		//texture reference assigned to the edge offsets
		size_t offset{0};
		hipBindTexture(&offset, tex_edgeOffsets, d_currQ, G.numVertices_m * sizeof(int));
	}

	/**
	 * The parent kernel is similar to launching the kernel from the cpu.
	 * But in this case launching the workload from the gpu itself.
	 */
	/*
	extern "C"
	__global__ void hierarchical::parentKernel(int *d_adjList,
			int *d_edgeOffsets,
			int *d_vertexDegree,
			int *d_distance,
			int *d_currQ,
			int *d_nextQ,
			int *d_nextQSize) {

		int currQSize = 1;
		int dev_depth = 0;
		int numBlocks;
		while (currQSize) {

			numBlocks = ((currQSize - 1) / BLOCK_SIZE) + 1;
			childKernel<<<numBlocks, BLOCK_SIZE>>>(++dev_depth, d_adjList, d_edgeOffsets,
					d_vertexDegree, d_distance,
					d_currQ, currQSize, d_nextQ, d_nextQSize);

			hipDeviceSynchronize(); // halt gpu
			currQSize = *d_nextQSize;
			hipMemcpyAsync(d_currQ, d_nextQ, sizeof(int) * currQSize, hipMemcpyDeviceToDevice);
			hipMemsetAsync(d_nextQSize, 0, sizeof(int));

		}
	}

	double hierarchical::executeDP(Graph &G, std::vector<int> &distanceCheck, int source) {

		//initialize data
		initMemory(G, source, distanceCheck);
		//execution
		auto start = std::chrono::high_resolution_clock::now();
		parentKernel<<<1, 1>>>(d_adjList, d_edgeOffsets, d_vertexDegree, d_distance,
								  d_currQ, d_nextQ, d_nextQSize);
		hipDeviceSynchronize();
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> t = end - start;
		//fill the distances obtained for comparison
		hipMemcpy(distanceCheck.data(), d_distance, G.numVertices_m * sizeof(int), hipMemcpyDeviceToHost);
		//free device pointers
		freeMemory();

		return t.count();
	}*/

	/**
	 *
	 * The kernel which performs the main operation of traversal over the graph.
	 */

	extern "C"
	__global__ void hierarchical::childKernel(int depth,
			int *d_adjList,
			int *d_edgeOffsets,
			int *d_vertexDegree,
			int *d_distance,
			int *d_currQ,
			int currQSize,
			int *d_nextQ,
			int *d_nextQSize) {

				/*
				 * d_variable is device allocated variable
				 * s_variable is shared memory variable
				 */
					__shared__ int s_subNextQ[NUM_SUB_QUEUES][SUB_QUEUE_SIZE], s_subNextQSize[NUM_SUB_QUEUES];
					__shared__ int s_globalOffsets[NUM_SUB_QUEUES];
					//registers
					int child, parent,
					subSharedQIdx /*which row of queue < NUM_SUB_QUEUES */,
					subSharedQSize/*length of a sub queue to be incremented < SUB_QUEUE_SIZE */,
					globalQIdx /*global level queue idx < |V| */;
					//obtain thread id
					int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
					if (threadIdx.x < NUM_SUB_QUEUES) //only one thread needed to set the size.
						s_subNextQSize[threadIdx.x] = 0;
					__syncthreads();

					if (tIdx < currQSize) {

						parent = d_currQ[tIdx];//get current values in parallel
						subSharedQIdx = tIdx & (NUM_SUB_QUEUES - 1);
//						subSharedQIdx = (tIdx / SUB_QUEUE_SIZE) % NUM_SUB_QUEUES;

						//expand all children
						for (int i=d_edgeOffsets[parent]; i<d_edgeOffsets[parent]+d_vertexDegree[parent]; ++i) {

							child = d_adjList[i];
							if (atomicMin(&d_distance[child], INT_MAX) == -1) { // if not found

								d_distance[child] = depth;
								// Increment sub queue size
								subSharedQSize = atomicAdd(&s_subNextQSize[subSharedQIdx], 1);
								if (subSharedQSize < SUB_QUEUE_SIZE) {
									s_subNextQ[subSharedQIdx][subSharedQSize] = child;

								}
								else {

									s_subNextQSize[subSharedQIdx] = SUB_QUEUE_SIZE;
									globalQIdx = atomicAdd(d_nextQSize, 1);
									d_nextQ[globalQIdx] = child;
								}
							}
						}
						__syncthreads();
					}

					if (threadIdx.x < NUM_SUB_QUEUES) // offsets for sub queues to global memory
							s_globalOffsets[threadIdx.x] = atomicAdd(d_nextQSize, s_subNextQSize[threadIdx.x]);
						__syncthreads();

					/*for (int t=threadIdx.x; t<SUB_QUEUE_SIZE; t+=blockDim.x) {

							for (int i=0; i<NUM_SUB_QUEUES; ++i) {
								if (t < s_subNextQSize[i]) {
									d_nextQ[s_globalOffsets[i] + t] = s_subNextQ[i][t];
								}
							}
					}*/


					for (int t=threadIdx.x; t<NUM_SUB_QUEUES*SUB_QUEUE_SIZE; t+=blockDim.x) {

						//row-major ordering lucky i guess
						const int row = t / SUB_QUEUE_SIZE;
						if (s_subNextQSize[row] == 0) continue;
						const int col = t % SUB_QUEUE_SIZE;
						int lim = (SUB_QUEUE_SIZE * row) + s_subNextQSize[row];
						if (t < lim)
							d_nextQ[s_globalOffsets[row] + col] = s_subNextQ[row][col];
					}
		}


	double hierarchical::execute(Graph &G, std::vector<int> &distanceCheck, int source) {

		//initialize data
		initMemory(G, source, distanceCheck);
		int currQSize{1};
		int depth{0}, numBlocks;
		//execution
		auto start = std::chrono::high_resolution_clock::now();
		while (currQSize) {
			numBlocks = ((currQSize - 1) / BLOCK_SIZE) + 1;
				childKernel<<<numBlocks, BLOCK_SIZE>>>(++depth, d_adjList, d_edgeOffsets,
						d_vertexDegree, d_distance,
						d_currQ, currQSize, d_nextQ, d_nextQSize);

				hipDeviceSynchronize(); // halt gpu
				hipMemcpyAsync(&currQSize, d_nextQSize, sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpyAsync(d_currQ, d_nextQ, sizeof(int) * currQSize, hipMemcpyDeviceToDevice);
				hipMemsetAsync(d_nextQSize, 0, sizeof(int));
		}
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> t = end - start;
		//fill the distances obtained for comparison
		hipMemcpy(distanceCheck.data(), d_distance, G.numVertices_m * sizeof(int), hipMemcpyDeviceToHost);
		//free device pointers
		freeMemory();

		return t.count();
	}

	extern "C"
	__global__ void blocked::kernelB(int depth, int *d_adjList, int *d_edgeOffsets,
				int *d_vertexDegree, int *d_distance, int *d_currQ, int currQSize,
				int *d_nextQ, int *d_nextQSize) {

			/*
			 * d_variable is device allocated variables
			 * s_variable is shared memory variable
			 */
			__shared__ int s_nextQ[BLOCK_QUEUE_SIZE];
			__shared__ int s_nextQSize, s_globalOffset;

			//obtain thread id
			int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
			if (tIdx == 0) //only one thread needed to set the size.
				s_nextQSize = 0;
			__syncthreads();

			if (tIdx < currQSize) {

				const int parent = d_currQ[tIdx];//get current values in parallel

				for (int i=d_edgeOffsets[parent]; i<d_edgeOffsets[parent]+d_vertexDegree[parent]; ++i) {

					const int child = d_adjList[i];
					if (atomicMin(&d_distance[child], INT_MAX) == -1) {

						d_distance[child] = depth;
						const int sharedQIdx = atomicAdd(&s_nextQSize, 1);

						if (sharedQIdx < BLOCK_QUEUE_SIZE) { //if the shared memory is not full, fill the shared queue

							s_nextQ[sharedQIdx] = child;
						}
						else { //fill the global queue

							s_nextQSize = BLOCK_QUEUE_SIZE;
							const int globalQIdx = atomicAdd(d_nextQSize, 1);
							d_nextQ[globalQIdx] = child;
						}
					}
				}
			}
			__syncthreads();

			if (threadIdx.x == 0) //offset for global memory
				s_globalOffset = atomicAdd(d_nextQSize, s_nextQSize);
			__syncthreads();

			if (threadIdx.x >= BLOCK_QUEUE_SIZE) return;
			for (int i=threadIdx.x; i<s_nextQSize; i+=blockDim.x) {// fill the global memory
				d_nextQ[s_globalOffset + i] = s_nextQ[i];
			}
		}

	double blocked::execute(Graph &G, std::vector<int> &distanceCheck, int source) {

		//initialize data
		initMemory(G, source, distanceCheck);
		int currQSize{1};
		int depth{0}, numBlocks;
		//execution
		auto start = std::chrono::high_resolution_clock::now();
		while (currQSize) {
			numBlocks = ((currQSize - 1) / BLOCK_SIZE) + 1;
				kernelB<<<numBlocks, BLOCK_SIZE>>>(++depth, d_adjList, d_edgeOffsets,
						d_vertexDegree, d_distance,
						d_currQ, currQSize, d_nextQ, d_nextQSize);

				hipDeviceSynchronize(); // halt gpu
				hipMemcpyAsync(&currQSize, d_nextQSize, sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpyAsync(d_currQ, d_nextQ, sizeof(int) * currQSize, hipMemcpyDeviceToDevice);
				hipMemsetAsync(d_nextQSize, 0, sizeof(int));
		}
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> t = end - start;
		//fill the distances obtained for comparison
		hipMemcpy(distanceCheck.data(), d_distance, G.numVertices_m * sizeof(int), hipMemcpyDeviceToHost);
		//free device pointers
		freeMemory();

		return t.count();
	}

	extern "C"
	__global__ void naive::kernelN(int depth, int *d_adjList, int *d_edgeOffsets,
			int *d_vertexDegree, int *d_distance, int *d_currQ, int currQSize,
			int *d_nextQ, int *d_nextQSize) {

				int t = threadIdx.x + blockDim.x * blockIdx.x;
				if (t < currQSize) {
					int parent = d_currQ[t];
					for (int i=d_edgeOffsets[parent]; i<d_edgeOffsets[parent]+d_vertexDegree[parent]; ++i) {
						int child = d_adjList[i];
						if (atomicMin(&d_distance[child], INT_MAX) == -1) {
							d_distance[child] = depth;
							int idx = atomicAdd(d_nextQSize, 1);
							d_nextQ[idx] = child;
						}
					}
				}
				__syncthreads();
	}

	double naive::execute(Graph &G, std::vector<int> &distanceCheck, int source) {

		//initialize data
		initMemory(G, source, distanceCheck);
		int currQSize{1};
		int depth{0}, numBlocks;
		//execution
		auto start = std::chrono::high_resolution_clock::now();
		while (currQSize) {
			numBlocks = ((currQSize - 1) / BLOCK_SIZE) + 1;
				kernelN<<<numBlocks, BLOCK_SIZE>>>(++depth, d_adjList, d_edgeOffsets,
						d_vertexDegree, d_distance,
						d_currQ, currQSize, d_nextQ, d_nextQSize);

				hipDeviceSynchronize(); // halt gpu
				hipMemcpyAsync(&currQSize, d_nextQSize, sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpyAsync(d_currQ, d_nextQ, sizeof(int) * currQSize, hipMemcpyDeviceToDevice);
				hipMemsetAsync(d_nextQSize, 0, sizeof(int));
		}
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> t = end - start;
		//fill the distances obtained for comparison
		hipMemcpy(distanceCheck.data(), d_distance, G.numVertices_m * sizeof(int), hipMemcpyDeviceToHost);
		//free device pointers
		freeMemory();

		return t.count();
	}

	void freeMemory(){

		hipFree(d_adjList);
		hipFree(d_edgeOffsets);
		hipFree(d_vertexDegree);
		hipFree(d_distance);
		hipFree(d_currQ);
		hipFree(d_nextQ);
		hipFree(d_nextQSize);
		hipUnbindTexture(tex_edgeOffsets);
	}

}

/*
 *
 * Testing texture memory
extern "C" __global__ void kernelTex(int *d_edgeOffsets) {

		int x = threadIdx.x;

		printf("\ntid(%d) = %d %d", x, tex1Dfetch(tex_edgeOffsets, x * 5), d_edgeOffsets[x * 5]);
	}

	double executeTex(Graph &G, std::vector<int> &distanceCheck, int source) {
		//assigning edge offsets to texture memory
		size_t offset;
		int arr[10] {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
		tex_edgeOffsets.addressMode[0] = hipAddressModeWrap;
		tex_edgeOffsets.filterMode = hipFilterModeLinear;
		tex_edgeOffsets.normalized = true;
		// Bind the array to the texture
		int *dev_ptr;
		hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
		gpuErrchk(hipMalloc(&dev_ptr, 10 * sizeof(int)));
		gpuErrchk(hipMemcpy(dev_ptr, arr, sizeof(int) * 10, hipMemcpyHostToDevice));

		gpuErrchk(hipBindTexture(&offset, tex_edgeOffsets, dev_ptr, 10 * sizeof(int)));
		gpuErrchk(hipBindTextureToArray(tex_edgeOffsets, dev_ptr, desc));

		initMemory(G, source, distanceCheck);
		kernelTex<<<1, 10>>>(d_edgeOffsets);
		hipDeviceSynchronize();
		freeMemory();
		hipUnbindTexture(tex_edgeOffsets);
		hipFree(dev_ptr);
		printf("%s", hipGetErrorString(hipGetLastError()));
	}
*/
