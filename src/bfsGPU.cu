#include "hip/hip_runtime.h"
#include "graph.hpp"
#include "bfsGPU.hpp"
#include <chrono>
#include <iostream>

#define BLOCK_SIZE 1024
#define BLOCK_QUEUE_SIZE 64
#define WARP_SIZE 32
#define NUM_SUB_QUEUES 4
#define NUM_SP 128 //CUDA cores per SM
/*
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;
hipFunction_t cuBFS;

#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	}																	\
}
*/


int * d_adjList;
int * d_edgeOffsets;
int * d_vertexDegree;
int * d_distance;
int * d_parent;
int * d_currQ;
int * d_nextQ;
int * d_currQSize;
int * d_nextQSize;



namespace bfsGPU {

	void initMemory(Graph &G, int source, std::vector<int> &distanceCheck) {

		//Initialize
		/*
		CHECK_CUDA_RESULT(hipInit(0));
		CHECK_CUDA_RESULT(hipDeviceGet(&cuDevice, 0));
		CHECK_CUDA_RESULT(hipCtxCreate(&cuContext, 0, cuDevice));
		CHECK_CUDA_RESULT(hipModuleLoad(&cuModule, "bfsGPU.ptx"));
		CHECK_CUDA_RESULT(hipModuleGetFunction(&cuBFS, cuModule, "kernelBfs"));*/
		//Allocation
		hipMalloc(&d_adjList, G.numEdges_m * sizeof(int));
		hipMalloc(&d_edgeOffsets, G.numVertices_m * sizeof(int));
		hipMalloc(&d_vertexDegree, G.numVertices_m * sizeof(int));
		hipMalloc(&d_distance, G.numVertices_m * sizeof(int));
		hipMalloc(&d_currQ, G.numVertices_m * sizeof(int));
		hipMalloc(&d_nextQ, G.numVertices_m * sizeof(int));
		hipMalloc(&d_currQSize, sizeof(int));
		hipMalloc(&d_nextQSize, sizeof(int));
		//Data transfer
		hipMemcpy(d_adjList, G.adjacencyList_m.data(), G.numEdges_m * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_edgeOffsets, G.edgeOffsets_m.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_vertexDegree, G.vertexDegree_m.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);
		//Init kernel parameters
		hipMemcpy(d_currQ, &source, sizeof(int), hipMemcpyHostToDevice);
		const int currQSize = 1;
		hipMemcpy(d_currQSize, &currQSize, sizeof(int), hipMemcpyHostToDevice);
		hipMemset(d_nextQSize, 0, sizeof(int));

		distanceCheck.resize(G.numVertices_m);
		std::fill(distanceCheck.begin(), distanceCheck.end(), -1);
		distanceCheck[source] = 0;
		hipMemcpy(d_distance, distanceCheck.data(), G.numVertices_m * sizeof(int), hipMemcpyHostToDevice);

	}

	extern "C" __global__ void kernelBfs(int depth, int *d_adjList, int *d_edgeOffsets,
				int *d_vertexDegree, int *d_distance, int *d_currQ, int *d_currQSize,
				int *d_nextQ, int *d_nextQSize) {

			/*
			 * d_variable is device allocated variables
			 * s_variable is shared memory variable
			 */
/*			int t = threadIdx.x + blockDim.x * blockIdx.x;

			if (t < *d_currQSize) {

				int parent = d_currQ[t];
				for (int i=d_edgeOffsets[parent]; i<d_edgeOffsets[parent]+d_vertexDegree[parent]; ++i) {

					int child = d_adjList[i];
					if (atomicMin(&d_distance[child], INT_MAX) == -1) {
						d_distance[child] = depth + 1;
						int idx = atomicAdd(d_nextQSize, 1);
						d_nextQ[idx] = child;
					}
				}
			}
			__syncthreads();*/
//			__shared__ int s_nextQ[BLOCK_QUEUE_SIZE];
			__shared__ int s_subNextQ[][], s_subQId;
			__shared__ int s_nextQSize, s_blockGlobalQIdx;

			//obtain thread id
			int tIdx = threadIdx.x + blockIdx.x * blockDim.x;
			if (tIdx == 0) //only one thread needed to set the size.
				s_nextQSize = 0;
			__syncthreads();

			if (tIdx < *d_currQSize) {

				const int parent = d_currQ[tIdx];//get current values in parallel

				for (int i=d_edgeOffsets[parent]; i<d_edgeOffsets[parent]+d_vertexDegree[parent]; ++i) {

					const int child = d_adjList[i];
					if (atomicMin(&d_distance[child], INT_MAX) == -1) {

						d_distance[child] = depth + 1;
						const int sharedQIdx = atomicAdd(&s_nextQSize, 1);

						if (sharedQIdx < BLOCK_QUEUE_SIZE) { //if the shared memory is not full, fill the shared queue

							s_nextQ[sharedQIdx] = child;
						}
						else { //fill the global queue

							s_nextQSize = BLOCK_QUEUE_SIZE;
							const int globalQIdx = atomicAdd(d_nextQSize, 1);
							d_nextQ[globalQIdx] = child;
						}
					}
				}
			}
			__syncthreads();

			if (threadIdx.x == 0) //offset for global memory
				s_blockGlobalQIdx = atomicAdd(d_nextQSize, s_nextQSize);
			__syncthreads();

			for (int i=threadIdx.x; i<s_nextQSize; i+=blockDim.x) {// fill the global memory
				d_nextQ[s_blockGlobalQIdx + i] = s_nextQ[i];
			}
		}

	double execute(Graph &G, std::vector<int> &distanceCheck, int source) {

		//initialize data
		initMemory(G, source, distanceCheck);
		//execution

		int h_currQSize{1};
		int numBlocks{0}, depth{0};

		auto start = std::chrono::high_resolution_clock::now();
		while (h_currQSize) {

			numBlocks = ((h_currQSize - 1) / BLOCK_SIZE) + 1;
			kernelBfs<<<numBlocks, BLOCK_SIZE>>>(depth, d_adjList, d_edgeOffsets, d_vertexDegree, d_distance,
					d_currQ, d_currQSize, d_nextQ, d_nextQSize);
			hipDeviceSynchronize(); // halt cpu
			std::swap(d_currQ, d_nextQ);
			hipMemcpy(d_currQSize, d_nextQSize, sizeof(int), hipMemcpyDeviceToDevice);
			hipMemset(d_nextQSize, 0, sizeof(int));
			hipMemcpy(&h_currQSize, d_currQSize, sizeof(int), hipMemcpyDeviceToHost);
			depth ++;

		}

		auto end = std::chrono::high_resolution_clock::now();

//		std::cout << hipGetErrorString(hipGetLastError())<< std::endl;
		std::chrono::duration<double, std::milli> t = end - start;
		//fill the distances obtained for comparison
		hipMemcpy(distanceCheck.data(), d_distance, G.numVertices_m * sizeof(int), hipMemcpyDeviceToHost);
		//free device pointers
		freeMemory();

		return t.count();
	}

	void freeMemory(){

		hipFree(d_adjList);
		hipFree(d_edgeOffsets);
		hipFree(d_vertexDegree);
		hipFree(d_distance);
		hipFree(d_parent);
		hipFree(d_currQ);
		hipFree(d_nextQ);
		hipFree(d_currQSize);
		hipFree(d_nextQSize);
	}
}
